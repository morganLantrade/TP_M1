#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

//Déclaration des prototypes des fonctions du fichier.
void reduce(unsigned int *vec,unsigned int*sum, int size);
__global__ void kreduce (unsigned int *tab, int size);


int main(int argc, char **argv){
  
  if (argc < 2){
    //Argument non valide
     printf("Usage: <filename>\n");
     exit(-1);
   }
   int size;
   unsigned int *vec;

   //On ouvre le fichier qu'on "place" dans f 
   FILE *f = fopen(argv[1],"r");
  //On récupère sa taille
   fscanf(f,"%d\n",&size);
   size = 1 << size;
   if (size >= (1 << 20)){
     printf("Size (%u) is too large: size is limited to 2^20\n",size);
     exit(-1);
   }
    //allocation de l'espace nécessaire au tableau vec
  vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);

  //boucle qui lit tous les éléments du fichier et les met dans vec
   for (int i=0; i<size; i++){
     fscanf(f, "%u\n",&(vec[i]));
   }
   unsigned int sum=0;

  reduce(vec,&sum,size);
  printf("sum = %u\n", sum);


  /*
  Code séquentiel basique qui fait la somme du vecteur de façon classique (permet de vérifier)
  unsigned int sum2 = 0;
  for (int i=0; i<size; i++){
    sum2 += vec[i];
  }
  printf("sum2 = %u\n", sum2);
  */
  fclose(f);
  return 0;
}

void reduce(unsigned int *vec,unsigned int *sum, int size){

  unsigned int *d_vec;
  int bytes = size*sizeof(unsigned int);
  //Allocation de l'espace mémoire du gpu pour stocker le tableau vecteur
  hipMalloc((void **)&d_vec, bytes);
  //Copie des donnec dans le gpu
  hipMemcpy(d_vec,vec,bytes,hipMemcpyHostToDevice);

  kreduce<<<1,size>>>(d_vec,size);

  /*Copie du résultat :
  On fait un Memcpy de d_vec : d_vec est un tableau mais comme on met juste d_vec sans indice (d_vec[x])
  alors d_vec vaut le premier élément du tableau => d_vec = d_vec[0] 
  */
  hipMemcpy(sum,d_vec,sizeof(unsigned int),hipMemcpyDeviceToHost);
  hipFree(d_vec);
}

__global__ void kreduce (unsigned int *tab, int size){
  int id=threadIdx.x;
  for(int offset=size/2;offset>=0;offset=offset/2){
    if(id<offset && id+offset<size){
      tab[id]+=tab[id+i];
    }
    __syncthreads();

  }
}
