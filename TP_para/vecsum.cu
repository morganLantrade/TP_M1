
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 4



void reduce(unsigned int *in, unsigned int *out,int size);
__device__ void scanBlock (unsigned int *in,int size);
__global__ void reduce1 (unsigned int *d_in, unsigned int *d_tmp,int size);
__global__ void reduce2 (unsigned int *d_in, int size);

int main(int argc, char **argv){
  if (argc < 2){
     printf("Usage: <filename>\n");
     exit(-1);
   }
   int size;
   unsigned int *vec;
   FILE *f = fopen(argv[1],"r");
   fscanf(f,"%d\n",&size);
   size = 1 << size;
   if (size >= (1 << 30)){
     printf("Size (%u) is too large: size is limited to 2^20\n",size);
     exit(-1);
   }
    vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
   for (int i=0; i<size; i++){
     fscanf(f, "%u\n",&(vec[i]));
   }
   unsigned int sum=0;
   reduce(vec,&sum,size);
   printf("sum = %u\n", sum);

   unsigned int sum2 = 0;
   for (int i=0; i<size; i++){
     sum2 += vec[i];
   }
   printf("sum2 = %u\n", sum2);

   fclose(f);
   return 0;
}

void reduce(unsigned int *in,unsigned int *out, int size){

  unsigned int *d_in,*d_out,*d_tmp;

  int bytes = size*sizeof(unsigned int);
  int num_blocks = size / BLOCK_SIZE;
  if (size % BLOCK_SIZE) num_blocks ++;
  int bytes_block = num_blocks*sizeof(unsigned int);

  hipMalloc((void **)&d_in, bytes);
  hipMalloc((void **)&d_out, bytes) ;
  hipMalloc((void **)&d_tmp, bytes_block);


  hipMemcpy(d_in,in,bytes,hipMemcpyHostToDevice);

  reduce1<<<num_blocks,BLOCK_SIZE>>>(d_in,d_tmp,size);
  //PARTIE 3
  
  if (num_blocks>BLOCK_SIZE){ // On re réduit 
    // calcul des differents tailles de blocs
    unsigned int *d_tmp2;
    int num_blocks2 = num_blocks / BLOCK_SIZE;
    int bytes_block2 = num_blocks2*sizeof(unsigned int);
    printf("nb sous blocs : %d \nnb blocs  : %d \n"  , num_blocks,num_blocks2);
    hipMalloc((void **)&d_tmp2, bytes_block2);
    reduce1<<<num_blocks2,BLOCK_SIZE>>>(d_tmp,d_tmp2,num_blocks);
    reduce2<<<1,num_blocks2>>>(d_tmp2,num_blocks2);
    hipMemcpy(out,d_tmp2,sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipFree(d_tmp2);
  //PARTIE 2
  }else{
    reduce2<<<1,num_blocks>>>(d_tmp,num_blocks);
    hipMemcpy(out,d_tmp,sizeof(unsigned int),hipMemcpyDeviceToHost);
  }

  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_tmp);

}

__device__ void scanBlock (unsigned int *d_in,int size){
  int id=threadIdx.x;
  for(int i=size/2;i>=1;i>>=1){
    if(id<i && id+i<size){
      d_in[id]+=d_in[id+i];
    }
    __syncthreads();
    }
  

}

__global__ void reduce1 (unsigned int *d_in,unsigned int *d_tmp, int size ){
  int block_id = blockIdx.x;
  int offset = block_id * BLOCK_SIZE;
  int n = BLOCK_SIZE;
  if ( block_id == gridDim.x-1){   // dernier bloc ( si il n'est rempli )
    n = size-block_id*BLOCK_SIZE;
  }
  scanBlock(&(d_in[offset]),n);
  if (threadIdx.x== 0){
    d_tmp[block_id] = d_in[offset] ;
    # if __CUDA_ARCH__>=200
      printf("Block %d  -> %d \n", block_id,d_in[offset]);
    #endif

  }
}


__global__ void reduce2 (unsigned int *d_in,int size ){
  scanBlock(d_in,size);



}
